#include "hip/hip_runtime.h"
// include system
#include <cstdio>

// include CUDA
#include "hip/hip_runtime.h"
#include ""

// include project
#include "integrator_exception.h"
#include "rk4.h"
#include "util.h"

#define THREADS_PER_BLOCK	256

static hipError_t HandleError(hipError_t cudaStatus, const char *file, int line)
{
    if (hipSuccess != cudaStatus) {
        printf( "%s in %s at line %d\n", hipGetErrorString( cudaStatus ), file, line );
        return cudaStatus;
    }
	return cudaStatus;
}
#define HANDLE_ERROR(cudaStatus) (HandleError(cudaStatus, __FILE__, __LINE__))

#define	LAMBDA	1.0/10.0

ttt_t rk4::c[] =  {0.0, 1.0/2.0, 1.0/2.0, 1.0, 1.0};
var_t rk4::a[] =  {0.0, 1.0/2.0, 1.0/2.0, 1.0, 1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0};
var_t rk4::bh[] = {1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0, 0.0};
var_t rk4::b[] =  {1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0 -LAMBDA, LAMBDA};


// ytemp = y_n + a*fr, r = 2, 3, 4
static __global__
void calc_ytemp_for_fr_kernel(int_t n, var_t *ytemp, const var_t *y_n, const var_t *fr, var_t a)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid) {
		ytemp[tid] = y_n[tid] + a * fr[tid];
		tid += stride;
	}
}

static __global__
void calc_yHat_kernel(int_t n, var_t *y_hat, const var_t *y_n, const var_t *f1, const var_t *f2, const var_t *f3, const var_t *f4, var_t b0, var_t b1)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid) {
		y_hat[tid] = y_n[tid] + b0 * (f1[tid] + f4[tid]) + b1 * (f2[tid] + f3[tid]);
		tid += stride;
	}
}

static __global__
void calc_f4_sub_f5_kernel(int_t n, var_t *result, const var_t *f4, const var_t* f5)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid) {
		result[tid] = f4[tid] - f5[tid];
		tid += stride;
	}
}

void rk4::calculate_grid(int nData, int threads_per_block)
{
	int	nThread = std::min(threads_per_block, nData);
	int	nBlock = (nData + nThread - 1)/nThread;
	grid.x  = nBlock;
	block.x = nThread;
}

void rk4::call_calc_ytemp_for_fr_kernel(int r)
{
	for (int i = 0; i < f.get_order(); i++) {
		int n		= f.d_y[i].size();
		var_t *y_n	= f.d_y[i].data().get();
		var_t *fr	= d_f[i][r-1].data().get();

		calculate_grid(f.d_y[i].size(), THREADS_PER_BLOCK);
		calc_ytemp_for_fr_kernel<<<grid, block>>>(n, d_ytemp[i].data().get(), y_n, fr, a[r] * dt_try);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus) {
			throw integrator_exception("calc_ytemp_for_fr_kernel failed");
		}
	}
}

void rk4::call_calc_yHat_kernel()
{
	for (int i = 0; i < f.get_order(); i++) {
		int n = f.d_y[i].size();
		var_t *y_n   = f.d_y[i].data().get();
		var_t *y_Hat = f.d_yout[i].data().get();
		var_t *f1	 = d_f[i][0].data().get();
		var_t *f2	 = d_f[i][1].data().get();
		var_t *f3	 = d_f[i][2].data().get();
		var_t *f4	 = d_f[i][3].data().get();

		calculate_grid(n, THREADS_PER_BLOCK);
		calc_yHat_kernel<<<grid, block>>>(n, y_Hat, y_n, f1, f2, f3, f4, b[0] * dt_try, b[1] * dt_try);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus) {
			throw integrator_exception("calc_yHat_kernel failed");
		}
	}
}

void rk4::call_calc_f4_sub_f5_kernel()
{
	for (int i = 0; i < f.get_order(); i++) {
		int n = f.d_y[i].size();
		var_t *err = d_err[i].data().get();
		var_t *f4  = d_f[i][3].data().get();
		var_t *f5  = d_f[i][4].data().get();

		calculate_grid(n, THREADS_PER_BLOCK);
		calc_f4_sub_f5_kernel<<<grid, block>>>(n, err, f4, f5);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus) {
			throw integrator_exception("calc_k5_sub_k4_kernel failed");
		}
	}
}

rk4::rk4(ode& f, ttt_t dt, bool adaptive, var_t tolerance) :
		integrator(f, dt),
		adaptive(adaptive),
		tolerance(tolerance),
		d_f(f.get_order()),
		d_ytemp(f.get_order(), d_var_t()),
		d_err(f.get_order(), d_var_t())
{
	RKOrder = 4;
	r_max = adaptive ? RKOrder + 1 : RKOrder;
	int	forder = f.get_order();

	for (int i = 0; i < forder; i++) {
		d_ytemp[i].resize(f.d_y[i].size());
		if (adaptive) {
			d_err[i].resize(f.d_y[i].size());
		}
		d_f[i].resize(r_max);
		for (int r = 0; r < r_max; r++) {
			d_f[i][r].resize(f.d_y[i].size());
		}
	}
}

ttt_t rk4::step()
{
	int	forder = f.get_order();

	int r = 0;
	// Calculate f1 = f(tn, yn) = d_f[][0]
	ttt_t ttemp = f.t + c[r] * dt;
	for (int i = 0; i < forder; i++) {
		f.calculate_dy(i, r, ttemp, f.d_p, f.d_y, d_f[i][r]);
	}

	dt_try = dt;
	var_t max_err = 0.0;
	int iter = 0;
	do {
		dt_did = dt_try;
		// Calculate f2 = f(tn + c2 * dt, yn + a21 * dt * f1) = d_f[][1]
		// Calculate f3 = f(tn + c3 * dt, yn + a31 * dt * f2) = d_f[][2]
		// Calculate f4 = f(tn + c4 * dt, yn + a41 * dt * f3) = d_f[][3]
		for (r = 1; r < RKOrder; r++) {
			ttemp = f.t + c[r] * dt_try;
			call_calc_ytemp_for_fr_kernel(r);
			for (int i = 0; i < forder; i++) {
				f.calculate_dy(i, r, ttemp, f.d_p, d_ytemp, d_f[i][r]);
			}
		}

		// yHat_(n+1) = yn + dt*(1/6*f1 + 1/3*f2 + 1/3*f3 + 1/6*f4) + O(dt^5)
		// f.d_yout = yHat_(n+1)
		call_calc_yHat_kernel();

		if (adaptive) {
			r = 4;
			ttemp = f.t + c[r] * dt_try;
			// Calculate f5 = f(tn + c5 * dt,  yn + dt*(1/6*f1 + 1/3*f2 + 1/3*f3 + 1/6*f4)) = d_f[][4]
			for (int i = 0; i < forder; i++) {
				f.calculate_dy(i, r, ttemp, f.d_p, f.d_yout, d_f[i][r]);
			}
			// calculate: d_err = h(f4 - f5)
			call_calc_f4_sub_f5_kernel();
			max_err = fabs(dt_try*LAMBDA*std::max(max_vec(d_err[0]), max_vec(d_err[1])));
			dt_try *= 0.9 * pow(tolerance / max_err, 1.0/4.0);
		}
		iter++;
	} while (adaptive && max_err > tolerance);
	if (adaptive)
	{
		update_counters(iter);
	}
	// Set the next step size
	dt = dt_try;

	f.tout = f.t + dt_did;
	f.swap_in_out();

	return dt_did;
}

string rk4::get_name()
{
	return adaptive ? "a_optRungeKutta4" : "optRungeKutta4";
}

#undef LAMBDA
