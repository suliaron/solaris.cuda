// include system
#include <Windows.h>

// include project
#include "stop_watch.h"

stop_watch::stop_watch() :
	_start(0),
	_stop(0)
{
	hipEventCreate(&_cuda_start);
	hipEventCreate(&_cuda_stop);
}

stop_watch::~stop_watch()
{
	hipEventDestroy(_cuda_start);
	hipEventDestroy(_cuda_stop);
}

int64_t stop_watch::start()
{
	_start = GetTimeMicro64();
	return _start;
}

int64_t stop_watch::stop()
{
	_stop = GetTimeMicro64();
	return _stop;
}

int64_t stop_watch::get_ellapsed_time()
{
	return (_stop - _start);
}

void stop_watch::cuda_start()
{
	hipEventRecord(_cuda_start, 0);
}

void stop_watch::cuda_stop()
{
	hipEventRecord(_cuda_stop, 0);
	hipEventSynchronize(_cuda_stop);
}

var_t stop_watch::get_cuda_ellapsed_time()
{
	float elapsed = 0.f;
	hipEventElapsedTime(&elapsed, _cuda_start, _cuda_stop);
	return (var_t)elapsed;
}

int64_t stop_watch::GetTimeMicro64()
{
	/* Windows */
	FILETIME ft;
	LARGE_INTEGER li;

	/* Get the amount of 100 nano seconds intervals elapsed since January 1, 1601 (UTC) and copy it
	* to a LARGE_INTEGER structure. */
	GetSystemTimeAsFileTime(&ft);
	li.LowPart = ft.dwLowDateTime;
	li.HighPart = ft.dwHighDateTime;

	uint64_t ret = li.QuadPart;
	ret -= 116444736000000000LL; /* Convert from file time to UNIX epoch time. */
	ret /= 10; /* From 100 nano seconds (10^-7) to 1 microsecond (10^-6) intervals */

	return ret;
}
